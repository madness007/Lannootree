#include "hip/hip_runtime.h"
#include <mean-masked.hpp>

__global__ void mean_of_mask(uchar3* d_input_image, u_int16_t* d_mask, uint3* d_total, uint* d_count, size_t rows, size_t cols) {
  size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= rows || y >= cols) return;
  
  uchar3 color = d_input_image[x * cols + y];

  atomicAdd(&d_total[d_mask[x * cols + y]].x, color.x);
  atomicAdd(&d_total[d_mask[x * cols + y]].y, color.y);
  atomicAdd(&d_total[d_mask[x * cols + y]].z, color.z);

  atomicAdd(&d_count[d_mask[x * cols + y]], 1);
}

__global__ void divide_total(uint3* d_total, uint* d_count, uint3* d_cstrign, int size) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= size) return;

  uint3 total_color = d_total[x];
  uint count = d_count[x];

  uint3 mean_color;
  mean_color.x = total_color.x / count;
  mean_color.y = total_color.y / count;
  mean_color.z = total_color.z / count;

  d_cstrign[x] = mean_color;
}

void mean_of_mask_launcher(uchar3* d_input_image, u_int16_t* d_mask, uint3* d_total, uint* d_count, size_t rows, size_t cols) {
  const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  uint grid_x = (uint) (rows / BLOCK_WIDTH + 1);
  uint grid_y = (uint) (cols / BLOCK_WIDTH + 1);
  const dim3 grid_size(grid_x, grid_y);

  mean_of_mask<<<grid_size, block_size>>>(d_input_image, d_mask, d_total, d_count, rows, cols);
}

void divide_total_launcher(uint3* d_total, uint* d_count, uint3* d_cstrign, int size) {
  const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  uint grid_x = (uint) (size / BLOCK_WIDTH + 1);
  const dim3 grid_size(grid_x, 1); 

  divide_total<<<grid_size, block_size>>>(d_total, d_count, d_cstrign, size);
}
