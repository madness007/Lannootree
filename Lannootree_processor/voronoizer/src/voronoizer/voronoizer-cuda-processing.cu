#include "hip/hip_runtime.h"
#include <voronoizer-cuda-processing.hpp>

uchar3* d_image = nullptr;
u_int16_t* d_mask = nullptr;
uint3* d_total = nullptr;
uint* d_count = nullptr;
uint3* d_cstring = nullptr;

void alloc_new_image_size(cv::Mat& image, cv::Mat& mask, size_t cstring_size) {
  cuda_check_errors(hipFree(d_image));
  cuda_check_errors(hipFree(d_mask));
  cuda_check_errors(hipFree(d_cstring));
  cuda_check_errors(hipFree(d_total));
  cuda_check_errors(hipFree(d_count));
  cuda_check_errors(hipFree(d_mask));

  cuda_check_errors(hipMalloc(&d_total, sizeof(uint3) * cstring_size));
  cuda_check_errors(hipMalloc(&d_count, sizeof(uint) * cstring_size));
  cuda_check_errors(hipMalloc(&d_image, sizeof(uchar3) * image.rows * image.cols));
  cuda_check_errors(hipMalloc(&d_mask, sizeof(u_int16_t) * image.rows * image.cols));
  cuda_check_errors(hipMalloc(&d_cstring, sizeof(uint3) * cstring_size));

  u_int16_t* maskPtr = mask.ptr<u_int16_t>(0);
  cuda_check_errors(hipMemcpy(d_mask, maskPtr, sizeof(u_int16_t) * image.rows * image.cols, hipMemcpyHostToDevice));
}

void process_image(cv::Mat& image, std::vector<uint3>& cstring) {
  cuda_check_errors(hipMemset(d_total, 0, sizeof(uint3) * cstring.size()));
  cuda_check_errors(hipMemset(d_count, 0, sizeof(uint) * cstring.size()));

  uchar3* imagePtr = (uchar3*) image.ptr<uchar>(0);
  cuda_check_errors(hipMemcpy(d_image, imagePtr, sizeof(uchar3) * image.rows * image.cols, hipMemcpyHostToDevice));


  mean_of_mask_launcher(d_image, d_mask, d_total, d_count, image.rows, image.cols);
  hipDeviceSynchronize();
  divide_total_launcher(d_total, d_count, d_cstring, cstring.size());
  hipDeviceSynchronize();

  cuda_check_errors(hipMemcpy(&cstring[0], d_cstring, sizeof(uint3) * cstring.size(), hipMemcpyDeviceToHost));
}
